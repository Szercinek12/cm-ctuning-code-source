#include "hip/hip_runtime.h"
/**
 * syr2k.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 *
 * Updated by Grigori Fursin (http://cTuning.org/lab/people/gfursin)
 * to work with Collective Mind Framework and OpenME interfqce for automatic 
 * and collective tuning and data mining: http://cTuning.org
 *
 */

#ifndef WINDOWS
 #include <unistd.h>
#endif

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>

#include <hip/hip_runtime.h>

#include "polybench.h"

#ifdef OPENME
#include <openme.h>
#endif

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

/* Problem size */
#define N 256 // 2048
#define M 256 // 2048

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 32
#define DIM_THREAD_BLOCK_Y 8

/* Declared constant values for ALPHA and BETA (same as values in PolyBench 2.0) */
#define ALPHA 12435
#define BETA 4546

/* Can switch DATA_TYPE between float and double */
# ifndef DATA_TYPE
#  define DATA_TYPE float
# endif

void init_arrays(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C)
{
	int i, j;
  
	for (i = 0; i < N; i++)
    	{
    		for (j = 0; j < N; j++)
		{
			C[i*N + j] = ((DATA_TYPE) i*j + 2) / N;
		}
      	
		for (j = 0; j < M; j++)
		{
	  		A[i*N + j] = ((DATA_TYPE) i*j) / N;
	  		B[i*N + j] = ((DATA_TYPE) i*j + 1) / N;
		}
    	}
}


void syr2k(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C)
{
	int i, j, k;
		
  	for (i = 0; i < N; i++)
	{
   		for (j = 0; j < N; j++)
		{
     			C[i*N + j] *= BETA;
		}
	}

  	for (i = 0; i < N; i++)
	{
   		for (j = 0; j < N; j++)
		{
      			for (k = 0; k < M; k++)
			{
	  			C[i*N + j] += ALPHA * A[i*M + k] * B[j*M + k];
	 		 	C[i*N + j] += ALPHA * B[i*M + k] * A[j*M + k];
			}
		}
	}
}


void compareResults(DATA_TYPE *C, DATA_TYPE *C_outputFromGpu)
{
	int i,j,fail;
	fail = 0;

	// Compare C with D
	for (i=0; i<N; i++)
	{
		for (j=0; j<N; j++)
		{
			if (percentDiff(C[i*N + j], C_outputFromGpu[i*N + j]) > PERCENT_DIFF_ERROR_THRESHOLD)
			{ 
				fail++;
			}
		}
	}
	
	// print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
  int devID = 0;
  hipError_t error;
  hipDeviceProp_t deviceProp;
  error = hipGetDevice(&devID);

  hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
  
  if (deviceProp.computeMode == hipComputeModeProhibited)
  {
    printf("Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
    exit(EXIT_SUCCESS);
  }

  if (error != hipSuccess)
    printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
  else
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);

  hipSetDevice( GPU_DEVICE );
}


__global__ void syr2k_kernel(DATA_TYPE *a, DATA_TYPE *b, DATA_TYPE *c)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < N) && (j < N))
	{
		c[i * N + j] *= BETA;
		
		int k;
		for(k = 0; k < M; k++)
		{
			c[i * N + j] += ALPHA * a[i * M + k] * b[j * M + k] + ALPHA * b[i * M + k] * a[j * M + k];
		}
	}
}


void syr2kCuda(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* C_outputFromGpu) 
{
        hipError_t error;
	double t_start, t_end;

	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;
	DATA_TYPE *C_gpu;

	error=hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * N * M);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * N * M);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMalloc((void **)&C_gpu, sizeof(DATA_TYPE) * N * N);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * N * M, hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * N * M, hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	error=hipMemcpy(C_gpu, C, sizeof(DATA_TYPE) * N * N, hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }
	
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid((size_t)ceil( ((float)N) / ((float)DIM_THREAD_BLOCK_X) ), (size_t)(ceil( ((float)N) / ((float)DIM_THREAD_BLOCK_Y) )));
	
//	t_start = rtclock();
	syr2k_kernel<<<grid,block>>>(A_gpu,B_gpu,C_gpu);
	hipDeviceSynchronize();
//	t_end = rtclock();
//	fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);
	
	error=hipMemcpy(C_outputFromGpu, C_gpu, sizeof(DATA_TYPE) * N * N, hipMemcpyDeviceToHost);
        if (error != hipSuccess)
        {
            printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

	hipFree(A_gpu);
	hipFree(B_gpu);
	hipFree(C_gpu);
}


int main()
{
  /* Prepare ctuning vars */
  long ct_repeat=0;
  long ct_repeat_max=1;

  double t_start, t_end;

  DATA_TYPE* A;
  DATA_TYPE* B;
  DATA_TYPE* C;
  DATA_TYPE* C_outputFromGpu;

#ifdef OPENME
  openme_init(NULL,NULL,NULL,0);
  openme_callback("PROGRAM_START", NULL);
#endif

  /* Run kernel. */
  if (getenv("CT_REPEAT_MAIN")!=NULL) ct_repeat_max=atol(getenv("CT_REPEAT_MAIN"));

  A = (DATA_TYPE*)malloc(N*M*sizeof(DATA_TYPE));
  B = (DATA_TYPE*)malloc(N*M*sizeof(DATA_TYPE));
  C = (DATA_TYPE*)malloc(N*M*sizeof(DATA_TYPE));
  C_outputFromGpu = (DATA_TYPE*)malloc(N*M*sizeof(DATA_TYPE));

  srand(1);
  init_arrays(A, B, C);
  GPU_argv_init();

#ifdef OPENME
  openme_callback("ACC_KERNEL_START", NULL);
#endif
  for (ct_repeat=0; ct_repeat<ct_repeat_max; ct_repeat++)
  {
    syr2kCuda(A, B, C, C_outputFromGpu);
  }
#ifdef OPENME
  openme_callback("ACC_KERNEL_END", NULL);
#endif

  srand(1);
  init_arrays(A, B, C);

#ifdef OPENME
  openme_callback("KERNEL_START", NULL);
#endif
  for (ct_repeat=0; ct_repeat<ct_repeat_max; ct_repeat++)
  {
    syr2k(A, B, C);
  }
#ifdef OPENME
  openme_callback("KERNEL_END", NULL);
#endif

  compareResults(C, C_outputFromGpu);

  free(A);
  free(B);
  free(C);
  free(C_outputFromGpu);

#ifdef OPENME
  openme_callback("PROGRAM_END", NULL);
#endif

  return 0;
}

